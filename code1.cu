
#include <hip/hip_runtime.h>
#include <stdio.h>
# define N 512

__global__ void add(int*a, int*b, int*c) {c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];}

int main(void) {

    int *a, *b, *c;
    int*d_a, *d_b, *d_c;    
    int size = N * sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int*)malloc(size); 
   //  random_ints(a, N);
    b = (int*)malloc(size); 
   //  random_ints(b, N);
    c = (int*)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<N,1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    printf("%d\n%d\n%d\n" , *c, *a,*b );

    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
    }